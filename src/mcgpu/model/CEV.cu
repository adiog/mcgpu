#include "hip/hip_runtime.h"
/*
 * Copyright 2012 Aleksander Gajewski <adiog@brainfuck.pl>
 *   created:  Thu 22 Mar 2012 01:17:43 PM CET
 *   modified: Wed Jun  6 09:33:04 2012
 */

#include <cmath>
#include "mcgpu/helpers/cuda_call.hpp"
#include "mcgpu/model/CEV.hpp"
#include "mcgpu/payoff/asian/Asian.hpp"
#include "mcgpu/payoff/european/European.hpp"
#include "mcgpu/simulation/Simulation.hpp"

__global__ void kernel_cev_eulermaruyama_european(
    float S0, float r, float sigma, float alpha, float T, int points,
    float *prices, unsigned int *rands, gpu_euro_apply apply,
    void *apply_args) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    float S = S0;
    float dt = T / points;

    hiprandStateXORWOW_t state;
    hiprand_init(rands[i], i, 0, &state);

    for (int t = 0; t < points; ++t) {
        S += r * S * dt +
             sigma * __powf(S, alpha) * sqrt(dt) * hiprand_normal(&state);
    }

    // store prices in global memory
    prices[i] = (*apply)(S, T, apply_args);
};

__global__ void kernel_cev_eulermaruyama_asian(
    float S0, float r, float sigma, float alpha, float T, int points,
    float *prices, unsigned int *rands, gpu_asian_apply apply, void *apply_args,
    gpu_asian_fold fold, void *fold_args, float init_acc) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    float S = S0;
    float acc = init_acc;
    float dt = T / points;

    hiprandStateXORWOW_t state;
    hiprand_init(rands[i], i, 0, &state);

    for (int t = 0; t < points; ++t) {
        S += r * S * dt +
             sigma * __powf(S, alpha) * sqrt(dt) * hiprand_normal(&state);
        acc = (*fold)(S, acc, dt * t, dt, fold_args);
    }

    // store prices in global memory
    prices[i] = (*apply)(S, acc, T, apply_args);
};

__global__ void kernel_cev_milstein_european(float S0, float r, float sigma,
                                             float alpha, float T, int points,
                                             float *prices, unsigned int *rands,
                                             gpu_euro_apply apply,
                                             void *apply_args) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    float S = S0;
    float dt = T / points;
    float dW;

    hiprandStateXORWOW_t state;
    hiprand_init(rands[i], i, 0, &state);

    for (int t = 0; t < points; ++t) {
        dW = sqrt(dt) * hiprand_normal(&state);
        S += r * S * dt + sigma * __powf(S, alpha) * dW +
             ((0.5 * (sigma * sigma) * alpha) * __powf(S, (2 * alpha - 1)) *
              ((dW * dW) - dt));
    }

    // store prices in global memory
    prices[i] = (*apply)(S, T, apply_args);
};

__global__ void kernel_cev_milstein_asian(float S0, float r, float sigma,
                                          float alpha, float T, int points,
                                          float *prices, unsigned int *rands,
                                          gpu_asian_apply apply,
                                          void *apply_args, gpu_asian_fold fold,
                                          void *fold_args, float init_acc) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    float S = S0;
    float acc = init_acc;
    float dt = T / points;
    float dW;

    hiprandStateXORWOW_t state;
    hiprand_init(rands[i], i, 0, &state);

    for (int t = 0; t < points; ++t) {
        dW = sqrt(dt) * hiprand_normal(&state);
        S += r * S * dt + sigma * __powf(S, alpha) * dW +
             (0.5 * (sigma * sigma) * alpha * __powf(S, 2 * alpha - 1) *
              ((dW * dW) - dt));
        acc = (*fold)(S, acc, dt * t, dt, fold_args);
    }

    // store prices in global memory
    prices[i] = (*apply)(S, acc, T, apply_args);
};

namespace mcgpu {
namespace model {

void CEV::runEulerSimulation(
    const mcgpu::payoff::european::European *payoff,
    const mcgpu::simulation::Simulation *simulation) const {
    kernel_cev_eulermaruyama_european<<<simulation->get_blocks(),
                                        simulation->get_threads()>>>(
        S0, r, sigma, alpha, T, simulation->get_points(),
        simulation->get_gpu_array(), simulation->get_gpu_seeds(),
        payoff->get_apply(), payoff->get_apply_args());
}

void CEV::runEulerSimulation(
    const mcgpu::payoff::asian::Asian *payoff,
    const mcgpu::simulation::Simulation *simulation) const {
    kernel_cev_eulermaruyama_asian<<<simulation->get_blocks(),
                                     simulation->get_threads()>>>(
        S0, r, sigma, alpha, T, simulation->get_points(),
        simulation->get_gpu_array(), simulation->get_gpu_seeds(),
        payoff->get_apply(), payoff->get_apply_args(), payoff->get_fold(),
        payoff->get_fold_args(), payoff->get_init_acc());
}

void CEV::runMilsteinSimulation(
    const mcgpu::payoff::european::European *payoff,
    const mcgpu::simulation::Simulation *simulation) const {
    kernel_cev_milstein_european<<<simulation->get_blocks(),
                                   simulation->get_threads()>>>(
        S0, r, sigma, alpha, T, simulation->get_points(),
        simulation->get_gpu_array(), simulation->get_gpu_seeds(),
        payoff->get_apply(), payoff->get_apply_args());
}

void CEV::runMilsteinSimulation(
    const mcgpu::payoff::asian::Asian *payoff,
    const mcgpu::simulation::Simulation *simulation) const {
    kernel_cev_milstein_asian<<<simulation->get_blocks(),
                                simulation->get_threads()>>>(
        S0, r, sigma, alpha, T, simulation->get_points(),
        simulation->get_gpu_array(), simulation->get_gpu_seeds(),
        payoff->get_apply(), payoff->get_apply_args(), payoff->get_fold(),
        payoff->get_fold_args(), payoff->get_init_acc());
}
}
}
