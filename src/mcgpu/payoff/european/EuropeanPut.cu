#include "hip/hip_runtime.h"
/*
 * Copyright 2012 Aleksander Gajewski <adiog@brainfuck.pl>
 *   created:  Sat 24 Mar 2012 08:45:31 PM CET
 *   modified: Sat 12 Dec 2015 09:16:35 PM CET
 */

#include "mcgpu/payoff/european/EuropeanPut.hpp"
#include "mcgpu/helpers/cuda_call.hpp"

__device__ float european_put_apply(float stock, float T, void *data) {
    float K = ((EuropeanPutApplyArgs *)data)->K;
    return ((stock < K) ? (K - stock) : 0.0F);
};

__global__ void get_european_put_apply(gpu_euro_apply *apply_ptr) {
    *apply_ptr = european_put_apply;
}

namespace mcgpu {
namespace payoff {
namespace european {

EuropeanPut::EuropeanPut(float K) : applyArgs{K} {
    gpu_euro_apply *function_to_pointer_on_device_ptr;
    CUDA_CALL(hipMalloc((void **)&function_to_pointer_on_device_ptr,
                         sizeof(gpu_euro_apply)));
    get_european_put_apply<<<1, 1>>>(function_to_pointer_on_device_ptr);
    CUDA_CALL(hipMemcpy(&gpu_apply, function_to_pointer_on_device_ptr,
                         sizeof(gpu_euro_apply), hipMemcpyDeviceToHost));
    CUDA_CALL(hipFree(function_to_pointer_on_device_ptr));

    CUDA_CALL(
        hipMalloc((void **)&gpu_apply_args, sizeof(EuropeanPutApplyArgs)));

    CUDA_CALL(hipMemcpy(gpu_apply_args, &applyArgs,
                         sizeof(EuropeanPutApplyArgs), hipMemcpyHostToDevice));
}

EuropeanPut::~EuropeanPut() { CUDA_CALL(hipFree(gpu_apply_args)); }
}
}
}
