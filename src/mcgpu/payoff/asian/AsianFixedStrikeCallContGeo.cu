#include "hip/hip_runtime.h"
/*
 * Copyright 2012 Aleksander Gajewski <adiog@brainfuck.pl>
 *   created:  Sat 24 Mar 2012 08:44:54 PM CET
 *   modified: Fri 11 Dec 2015 07:21:29 PM CET
 */

#include <cmath>
#include "mcgpu/payoff/asian/AsianFixedStrikeCallContGeo.hpp"
#include "mcgpu/helpers/cuda_call.hpp"

__device__ float asian_call_apply(float stock, float acc, float T, void *data) {
    float K = ((float *)data)[0];
    float contgeoavg = exp((1.0 / T) * acc);
    return ((contgeoavg > K) ? (contgeoavg - K) : 0.0);
};

__global__ void get_asian_call_apply(gpu_asian_apply *apply_ptr) {
    *apply_ptr = asian_call_apply;
}

__device__ float asian_call_fold(float stock, float acc, float t, float dT,
                                 void *data) {
    return acc + log(stock) * dT;
};

__global__ void get_asian_call_fold(gpu_asian_fold *fold_ptr) {
    *fold_ptr = asian_call_fold;
}

namespace mcgpu {
namespace payoff {
namespace asian {

AsianFixedStrikeCallContGeo::AsianFixedStrikeCallContGeo(float K_) : K(K_) {
    gpu_asian_apply *function_to_pointer_on_device_ptr;
    CUDA_CALL(hipMalloc((void **)&function_to_pointer_on_device_ptr,
                         sizeof(gpu_asian_apply)));
    get_asian_call_apply<<<1, 1>>>(function_to_pointer_on_device_ptr);
    CUDA_CALL(hipMemcpy(&gpu_apply, function_to_pointer_on_device_ptr,
                         sizeof(gpu_asian_apply), hipMemcpyDeviceToHost));
    CUDA_CALL(hipFree(function_to_pointer_on_device_ptr));

    gpu_asian_fold *function_to_pointer_fold_on_device_ptr;
    CUDA_CALL(hipMalloc((void **)&function_to_pointer_fold_on_device_ptr,
                         sizeof(gpu_asian_fold)));
    get_asian_call_fold<<<1, 1>>>(function_to_pointer_fold_on_device_ptr);
    CUDA_CALL(hipMemcpy(&gpu_fold, function_to_pointer_fold_on_device_ptr,
                         sizeof(gpu_asian_fold), hipMemcpyDeviceToHost));
    CUDA_CALL(hipFree(function_to_pointer_fold_on_device_ptr));

    CUDA_CALL(hipMalloc((void **)&gpu_apply_args, 1 * sizeof(float)));
    init_acc = 0;

    gpu_fold_args = (void *)0;

    float cpu_apply_args[1];
    cpu_apply_args[0] = K;
    CUDA_CALL(hipMemcpy(gpu_apply_args, cpu_apply_args, 1 * sizeof(float),
                         hipMemcpyHostToDevice));
}

AsianFixedStrikeCallContGeo::~AsianFixedStrikeCallContGeo() {
    CUDA_CALL(hipFree(gpu_apply_args));
}
}
}
}
